#include "hip/hip_runtime.h"
#include "header.h"
#include "dijkstra_header.h"

int main(int argc, char *argv[])
{
	if (argc != 2)
	{
		printf("Input file not found\n");
		return 0;
	}
	Graph *gr = create_graph(argv[1]);
	printf("File Loaded!\n");
	Graph *d_gr, temp;
	temp.nodes = gr->nodes;
	temp.edges = gr->edges;
	
	hipMalloc(&d_gr, sizeof(Graph));
	hipMalloc(&temp.destination, gr->edges * sizeof(unsigned int));
	hipMalloc(&temp.weight, gr->edges * sizeof(unsigned int));
	hipMalloc(&temp.first_edge, gr->nodes * sizeof(unsigned int));
	hipMalloc(&temp.out_degree, gr->nodes * sizeof(unsigned int));
	
	hipMemcpy(temp.destination, gr->destination, gr->edges * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(temp.weight, gr->weight, gr->edges * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(temp.first_edge, gr->first_edge, gr->nodes * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(temp.out_degree, gr->out_degree, gr->nodes * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_gr, &temp, sizeof(Graph), hipMemcpyHostToDevice);
	
	unsigned int startNode, *d_startNode, *d_currentNode, currentNodeOutdegree;	//to be fixed
	
	printf("Set start node id (1-%u): ", gr->nodes);
	scanf("%u", &startNode);
	startNode--;

	hipMalloc(&d_startNode, sizeof(unsigned int));
	hipMalloc(&d_currentNode, sizeof(unsigned int));
	hipMemcpy(d_startNode, &startNode, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_currentNode, &startNode, sizeof(unsigned int), hipMemcpyHostToDevice);

	Dijkstra *dijkstra;
	hipMalloc(&dijkstra, gr->nodes * sizeof(Dijkstra));

	dim3 block(BLOCK_SIZE);  
    dim3 grid(FRACTION_CEILING(gr->nodes, BLOCK_SIZE));	

	init<<<grid, block>>>(dijkstra, d_gr, d_startNode);

	currentNodeOutdegree = gr->out_degree[startNode];

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	while(1)
	{
    	dim3 block2(BLOCK_SIZE);  
    	dim3 grid2(FRACTION_CEILING(currentNodeOutdegree, BLOCK_SIZE));
    	update<<<grid2, block2>>>(d_gr, dijkstra, d_currentNode);

    	//Find min from unfixed nodes using Cub
		void     *d_temp_storage = NULL;
		size_t   temp_storage_bytes = 0;
		Dijkstra *d_out, out;
		out.cost = UINT_MAX;
		out.flag = 0;
		hipMalloc(&d_out, sizeof(Dijkstra));
		struct CustomMin min_op;
		hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, dijkstra, d_out, gr->nodes, min_op, out);
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, dijkstra, d_out, gr->nodes, min_op, out);
		hipMemcpy(&out, d_out, sizeof(Dijkstra), hipMemcpyDeviceToHost);

		hipMemset(&dijkstra[out.id].flag, 1, sizeof(char));	//fix new node
		hipMemcpy(d_currentNode, &out.id, sizeof(unsigned int), hipMemcpyHostToDevice);
		
		if (out.cost == UINT_MAX)
			break;

    	currentNodeOutdegree = gr->out_degree[out.id];
    	hipFree(d_out);
    	hipFree(d_temp_storage);
	}
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for kernels: %.4f ms\n", time);

	hipFree(dijkstra);
	hipFree(d_startNode);
	hipFree(d_currentNode);
	hipFree(temp.destination);
	hipFree(temp.weight);
	hipFree(temp.first_edge);
	hipFree(temp.out_degree);
	destroy_graph(gr);
}
