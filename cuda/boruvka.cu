#include "header.h"
#include "boruvka_header.h"

int main(int argc, char *argv[])
{
	if (argc != 2)
	{
		printf("Input file not found\n");
		return 0;
	}
	Graph *gr = create_graph(argv[1]);
	printf("File Loaded!\n");
	Graph *d_gr, temp, *d_gr2, temp2;
	unsigned int *d_minedge, *d_color, *d_flag, *d_EPS, *d_first_edge_copy;
	unsigned int change, *d_change, s1, s2;
	temp.nodes = gr->nodes;
	temp.edges = gr->edges;
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	hipMalloc(&d_change, sizeof(unsigned int));
	hipMalloc(&d_gr, sizeof(Graph));
	hipMalloc(&temp.destination, gr->edges * sizeof(unsigned int));
	hipMalloc(&temp.weight, gr->edges * sizeof(unsigned int));
	hipMalloc(&temp.first_edge, gr->nodes * sizeof(unsigned int));
	hipMalloc(&temp.out_degree, gr->nodes * sizeof(unsigned int));
	
	hipMemcpy(temp.destination, gr->destination, gr->edges * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(temp.weight, gr->weight, gr->edges * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(temp.first_edge, gr->first_edge, gr->nodes * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(temp.out_degree, gr->out_degree, gr->nodes * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_gr, &temp, sizeof(Graph), hipMemcpyHostToDevice);
	
	do
	{
		hipMalloc(&d_minedge, temp.nodes * sizeof(unsigned int));
		hipMalloc(&d_color, temp.nodes * sizeof(unsigned int));
		hipMalloc(&d_flag, temp.nodes * sizeof(unsigned int));
		hipMalloc(&d_EPS, temp.nodes * sizeof(unsigned int));

	    dim3 block(BLOCK_SIZE);  
    	dim3 grid(FRACTION_CEILING(temp.nodes, BLOCK_SIZE));
		
		find_min<<<grid,block>>>(d_gr, d_minedge);
		mirrors_edge<<<grid,block>>>(d_gr, d_minedge);
		initialize_colors<<<grid,block>>>(d_gr, d_minedge, d_color);
		do
		{
			hipMemset(d_change, 0, sizeof(unsigned int));
			propagate_colors<<<grid,block>>>(d_gr, d_color, d_change);
			hipMemcpy(&change, d_change, sizeof(unsigned int), hipMemcpyDeviceToHost);
		} while (change);
		create_new_vertex_ids<<<grid,block>>>(d_gr, d_color, d_flag);
		
		// EPS on flag array using CUB Library
		// Determine temporary device storage requirements
		void     *d_temp_storage = NULL;
		size_t   temp_storage_bytes = 0;
		hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_flag, d_EPS, temp.nodes);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		// Run exclusive prefix sum
		hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_flag, d_EPS, temp.nodes);
		hipFree(d_temp_storage);
		
		hipMemcpy(&s1, &d_EPS[temp.nodes-1], sizeof(unsigned int), hipMemcpyDeviceToHost);

		test<<<1,1>>>(d_gr, 1);
		if (s1 <= 1)
		{
			break;
		}

		hipMemcpy(&temp2.nodes, &d_EPS[temp.nodes-1], sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMalloc(&d_gr2, sizeof(Graph));
		hipMalloc(&temp2.first_edge, temp2.nodes * sizeof(unsigned int));
		hipMalloc(&temp2.out_degree, temp2.nodes * sizeof(unsigned int));
		hipMemset(temp2.out_degree, 0, temp2.nodes * sizeof(unsigned int));
		hipMemcpy(d_gr2, &temp2, sizeof(Graph), hipMemcpyHostToDevice);
		
		count_edges<<<grid,block>>>(d_gr, d_gr2, d_color, d_EPS);
		
		// EPS on outdegree array, gives firstedge array using CUB Library
		// Determine temporary device storage requirements
		d_temp_storage = NULL;
		temp_storage_bytes = 0;
		hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, temp2.out_degree, temp2.first_edge, temp2.nodes);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		// Run exclusive prefix sum
		hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, temp2.out_degree, temp2.first_edge, temp2.nodes);
		hipFree(d_temp_storage);
		
		hipMemcpy(&s1, &temp2.out_degree[temp2.nodes-1], sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(&s2, &temp2.first_edge[temp2.nodes-1], sizeof(unsigned int), hipMemcpyDeviceToHost);
		temp2.edges = s1 + s2;
		hipMalloc(&temp2.destination, temp2.edges * sizeof(unsigned int));
		hipMalloc(&temp2.weight, temp2.edges * sizeof(unsigned int));
		hipMemcpy(d_gr2, &temp2, sizeof(Graph), hipMemcpyHostToDevice);
		
		//test<<<1,1>>>(d_gr2, 1);
		hipMalloc(&d_first_edge_copy, temp2.nodes * sizeof(unsigned int));
		hipMemcpy(d_first_edge_copy, temp2.first_edge, temp2.nodes * sizeof(unsigned int), hipMemcpyDeviceToDevice);

		insert_new_edges<<<grid,block>>>(d_gr, d_gr2, d_color, d_first_edge_copy, d_EPS);
		
		hipFree(d_first_edge_copy);
		hipFree(d_minedge);
		hipFree(d_color);
		hipFree(d_flag);
		hipFree(d_EPS);
		//swap start
		hipFree(temp.destination);
		hipFree(temp.weight);
		hipFree(temp.first_edge);
		hipFree(temp.out_degree);
		temp.nodes = temp2.nodes;
		temp.edges = temp2.edges;
		temp.destination = temp2.destination;
		temp.weight = temp2.weight;
		temp.first_edge = temp2.first_edge;
		temp.out_degree = temp2.out_degree;
		hipMemcpy(d_gr, &temp, sizeof(Graph), hipMemcpyHostToDevice);
		temp2.destination = NULL;
		temp2.weight = NULL;
		temp2.first_edge = NULL;
		temp2.out_degree = NULL;
		hipFree(d_gr2);
		//swap end
	} while(1);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for kernels: %.4f ms\n", time);
	
	hipFree(temp.destination);
	hipFree(temp.weight);
	hipFree(temp.first_edge);
	hipFree(temp.out_degree);
	hipFree(d_minedge);
	hipFree(d_color);
	hipFree(d_change);
	hipFree(d_flag);
	hipFree(d_EPS);
	hipFree(d_gr);
	
	destroy_graph(gr);
}
